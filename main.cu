#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>

#define CUDA_CHECK(api_call) \
    do { \
        hipError_t error_status = (api_call); \
        if (error_status != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ \
                      << " for call '" << #api_call << "': " \
                      << hipGetErrorString(error_status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__device__ double device_f(double x) {
    return std::cos(x) / (std::log(1 + std::sin(x)) * std::sin(1 + std::sin(x)));
}

__global__ void integrate_kernel(double a, double delta_x, int n_steps, double* partial_sums_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_steps) {
        double x_i = a + (idx + 0.5) * delta_x;
        partial_sums_d[idx] = device_f(x_i) * delta_x;
    }
}

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

int main() {
    double a = 1e-9;
    double b = M_PI - 1e-9;
    const int total_computational_units = 1000000;
    const int threads_per_block = 1024;

    double delta_x = (b - a) / total_computational_units;

    std::cout << "Integrating f(x) = cot(x) / (ln(1+sin(x)) * sin(1+sin(x)))" << std::endl;
    std::cout << "Interval: [" << a << ", " << b << "]" << std::endl;
    std::cout << "Total computational units (N): " << total_computational_units << std::endl;
    std::cout << "Delta x: " << delta_x << std::endl;

    size_t size = total_computational_units * sizeof(double);
    double *partial_sums_d;
    std::vector<double> partial_sums_h(total_computational_units);

    CUDA_CHECK(hipMalloc(&partial_sums_d, size));

    hipEvent_t start_event, stop_event;
    CUDA_CHECK(hipEventCreate(&start_event));
    CUDA_CHECK(hipEventCreate(&stop_event));

    int num_blocks = (total_computational_units + threads_per_block - 1) / threads_per_block;

    CUDA_CHECK(hipEventRecord(start_event));

    integrate_kernel<<<num_blocks, threads_per_block>>>(a, delta_x, total_computational_units, partial_sums_d);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipEventRecord(stop_event));
    CUDA_CHECK(hipEventSynchronize(stop_event));

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(partial_sums_h.data(), partial_sums_d, size, hipMemcpyDeviceToHost));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start_event, stop_event));

    double total_sum = 0.0;
    for (int i = 0; i < total_computational_units; ++i) {
        total_sum += partial_sums_h[i];
    }

    std::cout << std::fixed << std::setprecision(15);
    std::cout << "Integral result: " << total_sum << std::endl;
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    CUDA_CHECK(hipEventDestroy(start_event));
    CUDA_CHECK(hipEventDestroy(stop_event));

    CUDA_CHECK(hipFree(partial_sums_d));

    return 0;
}
